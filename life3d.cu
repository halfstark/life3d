/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24011013
 * 姓名: 程家骏
 * 邮箱: chengjiajun20@gmail.com
 ------------------------------------------------*/


#include <hip/hip_runtime.h>
#include <chrono>
#include <cstddef>
#include <cstdint>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
#define VERIFY
#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]
#define ATin(x, y, z) in[(x) * N * N + (y) * N + z]
#define ATgolden(x, y, z) golden[(x) * N * N + (y) * N + z]

using std::cin, std::cout, std::endl;
using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}

__global__ void population_dev(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    printf("dev res %d\n",result);
    // return result;
}

// 打印世界状态
void print_universe(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}
bool verify_universe(int N, char *universe, char* golden)
{
    // 仅在N较小(<= 32)时用于Debug
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z) != ATgolden(x, y, z)) {
                    printf("verify failed\n");
                    return false;
                }
            }
        }
    }
    printf("verify success\n");
    return true;
}
__device__ void print_universe_dev(int N, char *universe)
{
    // 仅在N较小(<= 32)时用于Debug
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    printf("0 ");
                    // cout << "O ";
                else
                    printf("* ");
            }
            printf("\n");
        }
        printf("\n");
    }
    // cout << "population: " << population(N, universe) << endl;
}
__global__ void life3d(char* in, char* out, int N) {
    int x = blockIdx.z;
    int y = blockIdx.y;
    int z = threadIdx.x;
    // printf("x y z: %d %d %d\n", x, y, z);
    // int index = x*N*N + y*N + z;
    // print_universe_dev(N, in);

    // for (int i = 0; i < 4; i++) {
    //     for (int j = 0; j < 4; j++) {
    //         int x1 = x*4 + i;
    //         int y1 = y;
    //         int z1 = z*4 + j;
    //         // int index = (z*4 + j) + y*N + (x*4 + i)*N*N;
    //         // printf("x1 %d %d %d\n", x1, y1, z1);
    int cnt = 0;
    for (int dx = -1; dx <= 1; dx++)
        for (int dy = -1; dy <= 1; dy++)
            for (int dz = -1; dz <= 1; dz++) {
                if (dx == 0 && dy == 0 && dz == 0) continue;
                    int nx = (x + dx + N) % N;
                    int ny = (y + dy + N) % N;
                    int nz = (z + dz + N) % N;
                        // printf("")
                    cnt += ATin(nx, ny, nz);
            }
            // printf("cnt: %d\n", cnt);
            if (ATin(x, y, z) && (cnt < 5 || cnt > 7))
                out[x * N * N + y * N + z] = 0;
            else if (!ATin(x, y, z) && cnt == 6)
                out[x * N * N + y * N + z] = 1;
            else
                out[x * N * N + y * N + z] = ATin(x, y, z);
}
// 核心计算代码，将世界向前推进T个时刻
void life3d_run(int N, char *universe, int T, char* device_universe,char* device_out)
{
    dim3 g = dim3(1, N, N);
    dim3 b = dim3(N, 1, 1);
    // cudaMemcpy(device_universe, universe, N*N*N*sizeof(char), cudaMemcpyHostToDevice);


    for (int i = 0; i < T; i+=2) {
        // population_dev<<<1,1>>>(N, device_universe);
        life3d<<<g, b>>>(device_universe, device_out, N);
        life3d<<<g, b>>>(device_out, device_universe, N);
        // population_dev<<<1,1>>>(N, device_out);
        // cudaMemcpy(device_universe, device_out, N*N*N, cudaMemcpyDeviceToDevice);
        // cudaMemcpy(device_out, device_universe, N*N*N, cudaMemcpyDeviceToDevice);
        // cudaMemcpy(universe, device_out, N*N*N, cudaMemcpyDeviceToDevice);
        // print_universe(N, universe);
    }

}

void life3D_golden(int N, char *universe, int T) {
    char *next = (char *)malloc(N * N * N);
    for (int t = 0; t < T; t++)
    {
        // outerloop: iter universe
        for (int x = 0; x < N; x++)
            for (int y = 0; y < N; y++)
                for (int z = 0; z < N; z++)
                {
                    // inner loop: stencil
                    int alive = 0;
                    for (int dx = -1; dx <= 1; dx++)
                        for (int dy = -1; dy <= 1; dy++)
                            for (int dz = -1; dz <= 1; dz++)
                            {
                                if (dx == 0 && dy == 0 && dz == 0)
                                    continue;
                                int nx = (x + dx + N) % N;
                                int ny = (y + dy + N) % N;
                                int nz = (z + dz + N) % N;
                                alive += AT(nx, ny, nz);
                            }
                    if (AT(x, y, z) && (alive < 5 || alive > 7))
                        next[x * N * N + y * N + z] = 0;
                    else if (!AT(x, y, z) && alive == 6)
                        next[x * N * N + y * N + z] = 1;
                    else
                        next[x * N * N + y * N + z] = AT(x, y, z);
                }
        memcpy(universe, next, N * N * N);
    }
    free(next);
}

// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];

    char *universe = (char *)malloc(N * N * N);
    char *universe_golen = (char *)malloc(N * N * N);

    read_file(input_file, universe);
    memcpy(universe_golen, universe, N * N * N);
    #ifdef VERIFY
    life3D_golden(N, universe_golen, T);
    #endif
    hipError_t cudaStatus;

    char* dev_universe = NULL, *dev_out = NULL;
    cudaStatus = hipMalloc((void**)&dev_universe, N*N*N);
    if (cudaStatus != hipSuccess) {
        cout << "malloc failed\n";
        return -1;
    }
    cudaStatus = hipMalloc((void**)&dev_out, N*N*N);
    if (cudaStatus != hipSuccess) {
        cout << "malloc failed\n";
        return -1;
    }
    hipMemcpy(dev_universe, universe, N*N*N, hipMemcpyHostToDevice);
    // population_dev<<<1, 1>>>(N ,dev_universe);
    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    life3d_run(N, universe, T, dev_universe, dev_out);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    hipMemcpy(universe, dev_universe, N*N*N, hipMemcpyDeviceToHost);
    hipFree(dev_out);
    hipFree(dev_universe);
    #ifdef VERIFY
    verify_universe(N, universe, universe_golen);
    #endif
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);
    
    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    free(universe);
    return 0;
}
